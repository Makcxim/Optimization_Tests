#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <chrono>

// __device__ функция для проверки простоты числа
__device__ bool is_prime(uint64_t n) {
    if (n < 2) return false;
    for (uint64_t i = 2; i <= sqrt((double)n); ++i) {
        if (n % i == 0) return false;
    }
    return true;
}

// __device__ функция для проверки, имеет ли число ровно 5 делителей
__device__ bool is_needed(uint64_t n) {
    double sqrt_root = pow((double)n, 1.0 / 4.0);
    if (floor(sqrt_root) == sqrt_root && is_prime((uint64_t)sqrt_root)) {
        return true;
    }
    return false;
}

// __global__ функция-ядро для поиска чисел и записи результатов
__global__ void find_primes(uint64_t adjusted_start, uint64_t total_steps, uint64_t *results, uint64_t *result_count) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = gridDim.x * blockDim.x;

    for (uint64_t i = idx; i < total_steps; i += stride) {
        uint64_t n = adjusted_start + i * 240;
        if (is_needed(n)) {
            // Атомарное увеличение счетчика результатов
            uint64_t index = atomicAdd(result_count, 1);
            if (index < 500) { // Проверка на переполнение массива
                results[index] = n;
            }
        }
    }
}

int main() {
    uint64_t start = 1'000'000'000;
    uint64_t end = 17'000'000'000;

    // Оценка максимального числа результатов
    const int max_results = 500;

    uint64_t *d_results;
    uint64_t *d_result_count;

    // Выравнивание начала диапазона
    uint64_t adjusted_start = start - start % 240 + 1;
    uint64_t total_steps = (end - adjusted_start) / 240;

    // Выделение памяти на устройстве
    hipMalloc(&d_results, max_results * sizeof(uint64_t));
    hipMalloc(&d_result_count, sizeof(uint64_t));

    // Инициализация счетчика результатов нулем
    uint64_t zero = 0;
    hipMemcpy(d_result_count, &zero, sizeof(uint64_t), hipMemcpyHostToDevice);

    // Настройка параметров запуска ядра
    int threads_per_block = 256;
    int blocks_per_grid = 256;

    auto start_time = std::chrono::high_resolution_clock::now();

    // Запуск ядра CUDA
    find_primes<<<blocks_per_grid, threads_per_block>>>(adjusted_start, total_steps, d_results, d_result_count);

    hipDeviceSynchronize();

    auto end_time = std::chrono::high_resolution_clock::now();

    // Копирование результатов обратно на хост
    uint64_t result_count;
    hipMemcpy(&result_count, d_result_count, sizeof(uint64_t), hipMemcpyDeviceToHost);

    if (result_count > max_results) {
        result_count = max_results;
    }

    uint64_t *h_results = new uint64_t[result_count];
    hipMemcpy(h_results, d_results, result_count * sizeof(uint64_t), hipMemcpyDeviceToHost);

    // Запись результатов в файл
    std::ofstream file("needed_cuda.txt", std::ios::app);
    for (uint64_t i = 0; i < result_count; ++i) {
        std::cout << h_results[i] << std::endl;
        file << h_results[i] << "\n";
    }
    file.close();

    std::chrono::duration<double> elapsed = end_time - start_time;
    std::cout << "Time: " << elapsed.count() << " seconds" << std::endl;

    // Очистка памяти
    hipFree(d_results);
    hipFree(d_result_count);
    delete[] h_results;

    // Ожидание перед закрытием программы
    std::cout << "Press Enter to exit..." << std::endl;
    std::cin.get();

    return 0;
}
